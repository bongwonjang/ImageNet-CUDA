#include "CudaErrorHandling.h"

/*
	- checkXXX function
		The 3 functions, checkCUDA, checkCUDNN, and checkCUBLAS, verify runtime errors.
		
	- checkXXX 함수
		checkCUDA, checkCUDNN, checkCUBLAS 3가지 함수는 런타임 에러를 확인해준다.
*/
void checkCUDA(hipError_t status, const char * file, int line)
{
	if (status != hipSuccess)
	{
		printf("%s %s %d\n", hipGetErrorString(status), file, line);
		exit(-1);
	}
}

void checkCUDNN(hipdnnStatus_t status, const char * file, int line)
{
	if (status != HIPDNN_STATUS_SUCCESS)
	{
		printf("%s %s %d\n", hipdnnGetErrorString(status), file, line);
		exit(-1);
	}
}

void checkCUBLAS(hipblasStatus_t status, const char * file, int line)
{
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		printf("%s %s %d\n", cublasGetErrorString(status), file, line);
		exit(-1);
	}
}

/*
	- CudaContext::CudaContext
		Create cublas and cudnn handlers for using cuBLAS and cuDNN APIs.
		We will name these handlers the 'CUDA context'.

	- CudaContext::CudaContext
		cuBLAS, cuDNN API를 사용하기 위한 cublas, cudnn handler를 생성.
		이러한 handler들을 'CUDA context'라고 이름을 붙이겠다.
*/
CudaContext::CudaContext()
{
	hipblasCreate(&cublas_handle_);
	checkCUDNN(hipdnnCreate(&cudnn_handle_), __FILE__, __LINE__);
}
CudaContext::~CudaContext()
{
	hipblasDestroy(cublas_handle_);
	checkCUDNN(hipdnnDestroy(cudnn_handle_), __FILE__, __LINE__);
}
